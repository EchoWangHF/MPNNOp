#include "hip/hip_runtime.h"
// freshman.h头文件
//#ifndef FRESHMAN_H
//#define FRESHMAN_H
// 检查返回结果hipError_t是否成功
#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}
//#endif

// SumArrays.cu源代码文件
#include <hip/hip_runtime.h> // cuda相关头文件
#include <stdio.h>
//#include "freshman.h"

void initialData(float* ip,int size)
{
  time_t t;
  srand((unsigned )time(&t));
  for(int i=0;i<size;i++)
  {
    ip[i]=(float)(rand()%100);
  }
}

void sumArraysCpu(float*a, float*b, float*c, int num) {
  for (int i = 0; i < num; ++i) {
    c[i] = a[i] + b[i];
  }
}

bool resultCheck(float* cpu, float* gpu, int num) {
  for (int i = 0; i < num; ++i) {
    if (cpu[i] != gpu[i]) {
      printf("FAILED!\n");
    }
  }
  printf("PASSED!\n");
}

__global__ void sumArraysCuda(float*a,float*b,float*res)
{
  int i=threadIdx.x;
  res[i]=a[i]+b[i];
}

int main(int argc,char **argv)
{
  int dev = 0;
  hipSetDevice(dev); // 当有多个GPU时，选定cuda设备，默认是0即第一个主GPU，多GPU时0,1,2以此类推

  int nElem=32;
  printf("Vector size:%d\n",nElem);
  int nByte=sizeof(float)*nElem;
  float *a_h=(float*)malloc(nByte);
  float *b_h=(float*)malloc(nByte);
  float *res_gpu=(float*)malloc(nByte);
  float *res_cpu = (float*)malloc(nByte);
  // memset(res_h,0,nByte);

  float *a_d,*b_d,*res_d;
  CHECK(hipMalloc((float**)&a_d,nByte));
  CHECK(hipMalloc((float**)&b_d,nByte));
  CHECK(hipMalloc((float**)&res_d,nByte));

  initialData(a_h,nElem);
  initialData(b_h,nElem);

  CHECK(hipMemcpy(a_d,a_h,nByte,hipMemcpyHostToDevice)); // 从主机内存拷贝到设备内存
  CHECK(hipMemcpy(b_d,b_h,nByte,hipMemcpyHostToDevice));

  dim3 block(nElem);
  dim3 grid(nElem/block.x);
  sumArraysCuda<<<grid,block>>>(a_d,b_d,res_d);
  printf("Execution configuration<<<%d,%d>>>\n",block.x,grid.x);
  CHECK(hipMemcpy(res_gpu,res_d,nByte,hipMemcpyDeviceToHost)); // 将结果从设备内存拷贝到主机内存！
  hipFree(a_d);
  hipFree(b_d);
  hipFree(res_d);

  sumArraysCpu(a_h, b_h, res_cpu, nElem);
  resultCheck(res_cpu, res_gpu, nElem);
  free(a_h);
  free(b_h);
  free(res_gpu);
  free(res_cpu);

  return 0;
}

