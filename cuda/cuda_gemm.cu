#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void gemm(const float *A, const float *B, float *C, int m, int n, int k) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < m && j < n) {
    float sum = 0.0f;
    for (int l = 0; l < k; ++l) {
      sum += A[i * k + l] * B[l * n + j];
    }
    C[i * n + j] = sum;
  }
}

int main() {
  int m, n, k;
  float *A, *B, *C;

  // Get matrix dimensions from user
  scanf("%d %d %d", &m, &n, &k);

  // Allocate memory for matrices on host
  A = (float *)malloc(m * k * sizeof(float));
  B = (float *)malloc(k * n * sizeof(float));
  C = (float *)malloc(m * n * sizeof(float));

  // Initialize matrices
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < k; ++j) {
      A[i * k + j] = rand() / (float)RAND_MAX;
    }
  }
  for (int i = 0; i < k; ++i) {
    for (int j = 0; j < n; ++j) {
      B[i * n + j] = rand() / (float)RAND_MAX;
    }
  }

  // Copy matrices to device
  hipMalloc((void **)&d_A, m * k * sizeof(float));
  hipMalloc((void **)&d_B, k * n * sizeof(float));
  hipMalloc((void **)&d_C, m * n * sizeof(float));
  hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice);

  // Launch kernel
  dim3 gridDim(m / 16, n / 16);
  dim3 blockDim(16, 16);
  gemm<<<gridDim, blockDim>>>(d_A, d_B, d_C, m, n, k);

  // Copy results back to host
  hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

  // Print results
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      printf("%f ", C[i * n + j]);
    }
    printf("\n");
  }

  // Free memory
  free(A);
  free(B);
  free(C);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return 0;
}
