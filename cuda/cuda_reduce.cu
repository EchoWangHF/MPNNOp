#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void reduce(const float *A, float *C, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;

  if (i < n) {
    float sum = A[i];
    for (int s = 1; s < blockDim.x; s *= 2) {
      if (tid % (2 * s) == 0) {
        sum += A[i + s];
      }
    }
    C[i] = sum;
  }
}

int main() {
  int n;
  float *A, *C;

  // Get matrix dimensions from user
  scanf("%d", &n);

  // Allocate memory for matrices on host
  A = (float *)malloc(n * sizeof(float));
  C = (float *)malloc(n * sizeof(float));

  // Initialize matrices
  for (int i = 0; i < n; ++i) {
    A[i] = rand() / (float)RAND_MAX;
  }

  // Copy matrices to device
  hipMalloc((void **)&d_A, n * sizeof(float));
  hipMalloc((void **)&d_C, n * sizeof(float));
  hipMemcpy(d_A, A, n * sizeof(float), hipMemcpyHostToDevice);

  // Launch kernel
  dim3 gridDim(n / 16, 1);
  dim3 blockDim(16, 1);
  reduce<<<gridDim, blockDim>>>(d_A, d_C, n);

  // Copy results back to host
  hipMemcpy(C, d_C, n * sizeof(float), hipMemcpyDeviceToHost);

  // Print results
  for (int i = 0; i < n; ++i) {
    printf("%f ", C[i]);
  }
  printf("\n");

  // Free memory
  free(A);
  free(C);
  hipFree(d_A);
  hipFree(d_C);

  return 0;
}
